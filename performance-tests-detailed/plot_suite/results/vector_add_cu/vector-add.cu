#include "hip/hip_runtime.h"
#define BENCH
#include "descend.cuh"

#define WG XX
#define THREADS XX

template<std::size_t n>
auto inplace_vector_add(
        descend::i32 * const ha_array,
        const descend::i32 * const hb_array
) -> void {
    const auto gpu = descend::gpu_device(0);
    auto a_array = descend::gpu_alloc_copy<descend::array<descend::i32, n>>(&gpu, &*ha_array);
    const auto b_array = descend::gpu_alloc_copy<descend::array<descend::i32, n>>(&gpu, &*hb_array);
    descend::exec<WG, THREADS>(&gpu, [] __device__ (
            descend::i32 * const p0,
            const descend::i32 * const p1) -> void {
        p0[blockIdx.x * THREADS + threadIdx.x] = p0[blockIdx.x * THREADS + threadIdx.x] + p1[blockIdx.x * THREADS + threadIdx.x];
        __syncthreads();
    }, &a_array, &b_array);
    descend::copy_to_host<descend::array<descend::i32, n>>(&a_array, ha_array);
}

descend::Benchmark benchmark{descend::BenchConfig({"inplace_vector_add"})};
auto main() -> int {
    auto ha_array = descend::HeapBuffer<descend::array<descend::i32, WG*THREADS>>(descend::create_array<WG * THREADS, descend::i32>(0));
    const auto hb_array = descend::HeapBuffer<descend::array<descend::i32, WG*THREADS>>(descend::create_array<WG * THREADS, descend::i32>(1));
    inplace_vector_add<WG*THREADS>(&ha_array, &hb_array);

    // for (size_t i = 0; i < WG*THREADS; i++) {
    //     if (ha_array[i] != 1) {
    //         std::cout << "At i = " << i << "Wrong number. Found " << ha_array[i] << " instead of 1.";
    //         exit(EXIT_FAILURE);
    //     }
    // }
    std::cout << benchmark.avg_to_csv();
    exit(EXIT_SUCCESS);
}
