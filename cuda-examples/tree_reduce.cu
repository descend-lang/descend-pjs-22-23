#include "hip/hip_runtime.h"
#include "descend.cuh"
__device__ auto red_add(descend::i32 *const fst_half,
                        descend::i32 *const snd_half) -> void {
  *fst_half = *fst_half + *snd_half;
}

template <std::size_t n> auto reduce(descend::i32 *const ha_array) -> void {
  auto gpu = descend::gpu_device(0);
  auto a_array =
      descend::gpu_alloc<descend::array<descend::i32, n>>(&gpu, &*ha_array);

  descend::exec<64, 1024>(
      &gpu,
      [] __device__(descend::i32 * global_failure,
                    descend::i32 *const p0) -> void {
        if (*global_failure != -1) {
          return;
        }
        __syncthreads();
        for (descend::i32 k = 512; k > 0; k = k / 2) {

          if (threadIdx.x < k) {
            p0[blockIdx.x * 1024 + threadIdx.x] =
                p0[blockIdx.x * 1024 + threadIdx.x] +
                p0[blockIdx.x * 1024 + threadIdx.x + k];
          }
          __syncthreads();
        }
      },
      &a_array);
  descend::copy_to_host<descend::array<descend::i32, n>>(&a_array, ha_array);
}


auto main() -> int {
    auto ha_array = descend::HeapBuffer<descend::array<descend::i32, 64*1024>>(1);
    reduce<64*1024>(&ha_array);

    // for (size_t i = 0; i < 64; i++) {
    //     if (h_output[i] != 1024) {
    //         std::cout << "At i = " << i << "Wrong number. Found " << h_output[i] << " instead of 1024.";
    //         exit(EXIT_FAILURE);
    //     }
    // }
    exit(EXIT_SUCCESS);
}