#include "hip/hip_runtime.h"
#include "descend.cuh"

#define WG XX
#define THREADS XX

__device__ auto red_add(descend::i32 *const fst_half,
                        descend::i32 *const snd_half) -> void {
  *fst_half = *fst_half + *snd_half;
}

template <std::size_t n> auto reduce(descend::i32 *const ha_array) -> void {
  auto gpu = descend::gpu_device(0);
  auto a_array =
      descend::gpu_alloc_copy<descend::array<descend::i32, n>>(&gpu, &*ha_array);

  descend::exec<WG, THREADS>(
      &gpu,
      [] __device__(
                    descend::i32 *const p0) -> void {
        for (descend::i32 k = THREADS / 2; k > 0; k = k / 2) {

          if (threadIdx.x < k) {
            p0[blockIdx.x * THREADS + threadIdx.x] =
                p0[blockIdx.x * THREADS + threadIdx.x] +
                p0[blockIdx.x * THREADS + threadIdx.x + k];
          }
          __syncthreads();
        }
      },
      &a_array);
  descend::copy_to_host<descend::array<descend::i32, n>>(&a_array, ha_array);
}


auto main() -> int {
    auto ha_array = descend::HeapBuffer<descend::array<descend::i32, WG*THREADS>>(1);
    reduce<WG*THREADS>(&ha_array);

    // for (size_t i = 0; i < 64; i++) {
    //     if (h_output[i] != 1024) {
    //         std::cout << "At i = " << i << "Wrong number. Found " << h_output[i] << " instead of 1024.";
    //         exit(EXIT_FAILURE);
    //     }
    // }
    exit(EXIT_SUCCESS);
}