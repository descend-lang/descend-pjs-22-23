#include "hip/hip_runtime.h"
#include "descend.cuh"

#define WG XX
#define THREADS XX

template<std::size_t n>
auto reduce_shared_mem(
        const descend::i32 * const ha_array,
        descend::i32 * const h_output
) -> void {
    const auto gpu = descend::gpu_device(0);
    const auto a_array = descend::gpu_alloc_copy<descend::array<descend::i32, n>>(&gpu, ha_array);
    auto out_array = descend::gpu_alloc_copy<descend::array<descend::i32, WG>>(&gpu, &*h_output);
    descend::exec<WG, THREADS>(&gpu, [] __device__ (
            const descend::i32 * const p0,
            descend::i32 * const p1) -> void {
        __shared__ descend::i32 tmp[THREADS];
        tmp[threadIdx.x] = p0[blockIdx.x * THREADS + threadIdx.x];
        __syncthreads();
        for (descend::i32 k = THREADS / 2; k > 0; k = k / 2) {
            if (threadIdx.x < k)
            {
                tmp[threadIdx.x] = tmp[threadIdx.x] + tmp[threadIdx.x + k];
            }

            __syncthreads();
        }

        if (threadIdx.x < 1)
        {
            p1[blockIdx.x * 1 + threadIdx.x] = tmp[threadIdx.x];
        }

        __syncthreads();
        ;
    }, &a_array, &out_array);
    descend::copy_to_host<descend::array<descend::i32, WG>>(&out_array, h_output);
}
auto main() -> int {
    const auto ha_array = descend::HeapBuffer<descend::array<descend::i32, WG*THREADS>>(1);
    auto h_output = descend::HeapBuffer<descend::array<descend::i32, WG>>(0);
    reduce_shared_mem<WG*THREADS>(&ha_array, &h_output);

    // for (size_t i = 0; i < WG; i++) {
    //     if (h_output[i] != THREADS) {
    //         std::cout << "At i = " << i << "Wrong number. Found " << h_output[i] << " instead of 1024.";
    //         exit(EXIT_FAILURE);
    //     }
    // }
    exit(EXIT_SUCCESS);
}
